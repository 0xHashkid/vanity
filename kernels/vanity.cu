#include "hip/hip_runtime.h"
#include <stdio.h>
#include "base58.h"
#include "vanity.h"
#include "sha256.h"

__device__ int done = 0;
__device__ unsigned long long count = 0;

__device__ bool d_case_insensitive = false;

// TODO:
// 1) Should maybe write a macro for the err handling
// 2) Theoretically possible to reuse device reallocs but it's only one per round so it's kind of ok
extern "C" void vanity_round(
    int id,
    uint8_t *seed,
    uint8_t *base,
    uint8_t *owner,
    char *target,
    uint64_t target_len,
    uint8_t *out,
    bool case_insensitive)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (id >= deviceCount)
    {
        printf("Invalid GPU index: %d\n", id);
        return;
    }

    // Set device and initialize it
    hipSetDevice(id);
    gpu_init(id);

    // Allocate device buffer for seed, base, owner, out, target len, target
    uint8_t *d_buffer;
    hipError_t err = hipMalloc(
        (void **)&d_buffer,
        32               // seed
            + 32         // base
            + 32         // owner
            + 8          // target len
            + target_len // target
            + 16         // out (16 byte seed)
    );
    if (err != hipSuccess)
    {
        printf("CUDA malloc error (d_buffer): %s\n", hipGetErrorString(err));
        return;
    }

    // Copy input seed, base, owner to device
    err = hipMemcpy(d_buffer, seed, 32, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (seed): %s\n", hipGetErrorString(err));
        return;
    }
    err = hipMemcpy(d_buffer + 32, base, 32, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (base): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }
    err = hipMemcpy(d_buffer + 64, owner, 32, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (owner): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }
    err = hipMemcpy(d_buffer + 96, &target_len, 8, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (target_len): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }
    err = hipMemcpy(d_buffer + 104, target, target_len, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (target): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }
    err = hipMemcpyToSymbol(HIP_SYMBOL(d_case_insensitive), &case_insensitive, 1, 0, hipMemcpyHostToDevice);

    // Reset tracker and counter using hipMemcpyToSymbol
    int zero = 0;
    unsigned long long zero_ull = 0;
    err = hipMemcpyToSymbol(HIP_SYMBOL(done), &zero, sizeof(int));
    if (err != hipSuccess)
    {
        printf("CUDA memcpy to symbol error (done): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }
    err = hipMemcpyToSymbol(HIP_SYMBOL(count), &zero_ull, sizeof(unsigned long long));
    if (err != hipSuccess)
    {
        printf("CUDA memcpy to symbol error (count): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }

    // Launch vanity search kernel
    vanity_search<<<num_blocks, num_threads>>>(d_buffer, num_blocks * num_threads);
    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA launch error: %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }

    // Copy result to host
    err = hipMemcpy(out, d_buffer + 104 + target_len, 16, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (d_out): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }
    err = hipMemcpyFromSymbol(out + 16, HIP_SYMBOL(count), 8, 0, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (count): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }

    // Free pointers
    hipFree(d_buffer);
}

__device__ uint8_t const alphanumeric[63] = "0123456789ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz";

__global__ void
vanity_search(uint8_t *buffer, uint64_t stride)
{
    // Deconstruct buffer
    uint8_t *seed = buffer;
    uint8_t *base = buffer + 32;
    uint8_t *owner = buffer + 64;
    uint64_t target_len;
    memcpy(&target_len, buffer + 96, 8);
    uint8_t *target = buffer + 104;
    uint8_t *out = (buffer + 104 + target_len);

    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned char local_out[32] = {0};
    unsigned char local_encoded[44] = {0};
    uint64_t local_seed[4];

    // Pseudo random generator
    CUDA_SHA256_CTX ctx;
    cuda_sha256_init(&ctx);
    cuda_sha256_update(&ctx, (BYTE *)(seed), 32);
    cuda_sha256_update(&ctx, (BYTE *)(&idx), 8);
    cuda_sha256_final(&ctx, (BYTE *)local_seed);

    CUDA_SHA256_CTX address_sha;
    cuda_sha256_init(&address_sha);
    cuda_sha256_update(&address_sha, (BYTE *)base, 32);

    for (uint64_t iter = 0; iter < 1000 * 1000 * 1000; iter++)
    {
        // Has someone found a result?
        if (iter % 100 == 0)
        {
            if (atomicMax(&done, 0) == 1)
            {
                atomicAdd(&count, iter);
                return;
            }
        }

        cuda_sha256_init(&ctx);
        cuda_sha256_update(&ctx, (BYTE *)local_seed, 16);
        cuda_sha256_final(&ctx, (BYTE *)local_seed);

        uint32_t *indices = (uint32_t *)&local_seed;
        uint8_t create_account_seed[16] = {
            alphanumeric[indices[0] % 62],
            alphanumeric[indices[1] % 62],
            alphanumeric[indices[2] % 62],
            alphanumeric[indices[3] % 62],
            alphanumeric[indices[4] % 62],
            alphanumeric[indices[5] % 62],
            alphanumeric[indices[6] % 62],
            alphanumeric[indices[7] % 62],
            alphanumeric[(indices[0] >> 2) % 62],
            alphanumeric[(indices[1] >> 2) % 62],
            alphanumeric[(indices[2] >> 2) % 62],
            alphanumeric[(indices[3] >> 2) % 62],
            alphanumeric[(indices[4] >> 2) % 62],
            alphanumeric[(indices[5] >> 2) % 62],
            alphanumeric[(indices[6] >> 2) % 62],
            alphanumeric[(indices[7] >> 2) % 62],
        };

        // Calculate and encode public
        CUDA_SHA256_CTX address_sha_local;
        memcpy(&address_sha_local, &address_sha, sizeof(CUDA_SHA256_CTX));
        cuda_sha256_update(&address_sha_local, (BYTE *)create_account_seed, 16);
        cuda_sha256_update(&address_sha_local, (BYTE *)owner, 32);
        cuda_sha256_final(&address_sha_local, (BYTE *)local_out);
        fd_base58_encode_32(local_out, (unsigned char *)(&local_encoded), d_case_insensitive);

        // Check target
        if (matches_target((unsigned char *)local_encoded, (unsigned char *)target, target_len))
        {
            // Are we first to write result?
            if (atomicMax(&done, 1) == 0)
            {
                // seed for CreateAccountWithSeed
                memcpy(out, create_account_seed, 16);
            }

            atomicAdd(&count, iter + 1);
            return;
        }
    }
}

__device__ bool matches_target(unsigned char *a, unsigned char *target, uint64_t n)
{
    for (int i = 0; i < n; i++)
    {
        if (a[i] != target[i])
            return false;
    }
    return true;
}

__device__ uint8_t hi[32] = {
    11, 173, 87, 184, 147, 234, 104, 132, 137, 81, 39, 28, 37, 108, 223, 35, 44, 157, 207, 159, 1, 96, 51, 104, 99, 171, 194, 255, 255, 255, 255, 255};
__device__ uint8_t lo[32] = {
    11, 173, 87, 184, 50, 176, 108, 7, 110, 59, 244, 93, 50, 24, 226, 112, 40, 115, 18, 145, 61, 45, 152, 223, 115, 133, 252, 192, 0, 0, 0, 0};
__device__ bool in_target_range(unsigned char *a)
{
    for (int i = 0; i < 32; i++)
    {
        if (a[i] == hi[i] && a[i] == lo[i])
            continue;
        if (a[i] <= hi[i] && a[i] >= lo[i])
            return true;
        else
            return false;
    }
    return false;
}
